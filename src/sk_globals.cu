
#include <hip/hip_runtime.h>
// Autogenerated by python -m sk_bias emit_code

namespace n2k {
namespace sk_globals {

// mu_min = 1.0;
// mu_max = 90.0;

double xmin = -0.03599847736264212;
double xmax = 4.535808147692907;
int nx = 128;
int ny = 4;
int n_min = 64;

float bias_coeffs[512] = {
  0.0018437357020770762,  0.4234818333361899,  3.08423585770428,  19.605219754738016,
  -0.0019930847752644976,  0.3432293516497397,  2.558259212230691,  18.378343882520806,
  -0.004758143762124911,  0.27655521360487245,  2.083826035501058,  16.94960132117438,
  -0.006665268882807052,  0.22191078799265893,  1.6643220596547612,  15.363975965839527,
  -0.007899551873285171,  0.17771791008622984,  1.3009133039745406,  13.678925679850936,
  -0.008617622140482561,  0.14243153521418245,  0.9926457323445089,  11.95702580218461,
  -0.008949128692732788,  0.1145913579905766,  0.7367249769856729,  10.259316348397899,
  -0.008999070950063581,  0.09286023773174885,  0.5289186113027495,  8.639810020162022,
  -0.008850643578908635,  0.07604904666858657,  0.3640099656952339,  7.1417953480121685,
  -0.008568308216489351,  0.06312881631521705,  0.23625057657000142,  5.795917623836387,
  -0.008200863776880007,  0.05323199405103195,  0.13976986671537256,  4.619776176361316,
  -0.007784347962488836,  0.04564517253887247,  0.06890605798434754,  3.6190565788566436,
  -0.007344659482175997,  0.03979569691871986,  0.018458337763606024,  2.789225874250329,
  -0.006899839628643711,  0.03523463755955801,  -0.016161359455052034,  2.118178576158129,
  -0.006461991475778639,  0.031618057693577846,  -0.03884759658558339,  1.5886374202673232,
  -0.006038844610107841,  0.028688328109414133,  -0.05279377814736073,  1.1806192343355855,
  -0.005634993567438462,  0.02625660694538467,  -0.06052916551288835,  0.8733588279137166,
  -0.005252850165580636,  0.024187340779199932,  -0.06399688091641975,  0.6470668281835669,
  -0.004893355239351749,  0.022385040727375127,  -0.06463729600045538,  0.4837497491243986,
  -0.004556495518687198,  0.020783578207627336,  -0.06348941469582443,  0.3680864350026809,
  -0.004241668146079958,  0.019337695702102305,  -0.0612727020322823,  0.287335347878922,
  -0.003947930002634159,  0.018016638577435636,  -0.05847261699318647,  0.23152524686888043,
  -0.0036741627632335128,  0.016799439743518715,  -0.05540243855671119,  0.19301608734528716,
  -0.003419178282271229,  0.015671588781241752,  -0.05225544497898169,  0.16614514916723644,
  -0.0031817830980043205,  0.014622785730232126,  -0.04915086852770387,  0.14709356875390533,
  -0.002960815867538491,  0.013645336165751587,  -0.04614784913883292,  0.1329392725601499,
  -0.002755167511439671,  0.012733287099845109,  -0.04328373558450821,  0.12205982627340471,
  -0.002563790764914531,  0.011881713448779863,  -0.04056944775334103,  0.11317289182621745,
  -0.0023857035780490143,  0.01108644482429355,  -0.03801167754727399,  0.1056585419573634,
  -0.0022199892601915128,  0.01034383177926656,  -0.03561192437132308,  0.09915634382920925,
  -0.0020657952620253273,  0.009650656989769857,  -0.033369741074693406,  0.09345526547985335,
  -0.0019223319137249284,  0.00900415984974501,  -0.03128906518736633,  0.08854692788738266,
  -0.0017888721736600832,  0.008402117338306427,  -0.029382122840616105,  0.08463444683002853,
  -0.00166475341240635,  0.007842993036742742,  -0.027671036191350206,  0.08203498383227095,
  -0.0015493823960139874,  0.00732620692597763,  -0.026195091978933948,  0.08130245657228581,
  -0.001442244894119371,  0.0068525138666863964,  -0.025016220817861052,  0.0832157816912588,
  -0.0013429216759055507,  0.006424514984761488,  -0.024224058479830904,  0.08872425700245738,
  -0.0012511130225727474,  0.006047471598588316,  -0.023961600807978992,  0.09960367185702387,
  -0.00116667422715655,  0.005730080383248295,  -0.024413770831643367,  0.11764903858421899,
  -0.0010896648037141965,  0.0054856944952597075,  -0.025832803167759287,  0.14522898728319128,
  -0.0010204142212724143,  0.005333678790190534,  -0.0285505995805126,  0.1853421189298436,
  -0.0009596068430706151,  0.005300955913746464,  -0.03298757170500742,  0.24151687503036737,
  -0.0009083883230884826,  0.00542364449649213,  -0.03964987239987761,  0.3173459353420243,
  -0.0008684949472077541,  0.005748986558948843,  -0.04915623055353531,  0.417275344327206,
  -0.0008424062870189264,  0.006336871234111641,  -0.06219744774102756,  0.5449403208185631,
  -0.0008335200773730289,  0.007261701851444259,  -0.07956845641706424,  0.7045375570887652,
  -0.0008463464965962064,  0.008613484747936441,  -0.10211648161839533,  0.8992486547649509,
  -0.0008867171189919087,  0.010498772316775984,  -0.13072905581020683,  1.131473335044577,
  -0.0009620018624092452,  0.013040876995422222,  -0.16629639847360436,  1.4024132821795785,
  -0.0010813254320667376,  0.016379279469234124,  -0.20965840689950743,  1.7113718071778,
  -0.0012557732400553245,  0.020668355335111244,  -0.26157616620412166,  2.0562758361920364,
  -0.001498575727665541,  0.026074803418037362,  -0.32264220750653266,  2.4320306948502415,
  -0.0018252595759258305,  0.032774662841660536,  -0.39327744020533917,  2.832310990270375,
  -0.00225375455924004,  0.04094875710958236,  -0.47362742861992346,  3.247542795827136,
  -0.002804445820793422,  0.05077798966676283,  -0.5635688302293124,  3.6669356997734797,
  -0.003500163105456089,  0.062437564058541284,  -0.6626553559360655,  4.077941887240633,
  -0.0043661008841979765,  0.0760908227868633,  -0.77008759347336,  4.466354069285266,
  -0.005429666211521189,  0.09188321378879749,  -0.8847388554775083,  4.817944131078516,
  -0.006720254358292044,  0.10993597285837711,  -1.0051380630809492,  5.118024488445361,
  -0.008268955567122926,  0.13034058305250085,  -1.1295268565538672,  5.353188414694092,
  -0.010108199435561849,  0.1531535265260686,  -1.255873242423824,  5.510851982704125,
  -0.01227134624590609,  0.1783924604753722,  -1.381978701545679,  5.581745461226095,
  -0.014792236852012808,  0.20603274730454058,  -1.5054771887225917,  5.557832309132623,
  -0.017704714382386966,  0.23600611087671908,  -1.623979152704795,  5.435291489440176,
  -0.02104213191650639,  0.2681999022724178,  -1.7351069837329822,  5.213137318496799,
  -0.024836860493409496,  0.3024581145319978,  -1.8365975897322602,  4.8943144024960334,
  -0.02911981128874097,  0.3385833961322753,  -1.9263476838518663,  4.484375682091529,
  -0.03391998466560153,  0.3763405240711607,  -2.002506372745767,  3.9924061127532844,
  -0.039264057193182156,  0.41546083897846847,  -2.063540026224923,  3.4309294050904873,
  -0.045176015774920586,  0.45564708068211607,  -2.108217856686239,  2.8132191870688206,
  -0.05167684580363607,  0.49657951754350915,  -2.135691862170699,  2.1546930310581986,
  -0.058784278074387464,  0.5379221150369986,  -2.145503742368882,  1.4720282966446883,
  -0.06651259691313816,  0.5793288538835846,  -2.1375687730526276,  0.7817654582900391,
  -0.07487250998289574,  0.6204501756647645,  -2.1121812273881906,  0.1005737704106734,
  -0.08387107837681795,  0.660938939901687,  -2.0699482358238717,  -0.5571882960207676,
  -0.09351170406520715,  0.7004564382519977,  -2.0118043301061355,  -1.1777800028048666,
  -0.1037941705096892,  0.7386776516010303,  -1.938955340604722,  -1.7493859848032964,
  -0.11471473134370891,  0.7752959311399826,  -1.8528444394223622,  -2.2616387860765705,
  -0.12626624134655762,  0.8100266241640776,  -1.7550396552008198,  -2.7083583129852045,
  -0.13843832362192243,  0.8426110327792388,  -1.647293420315696,  -3.0834563769755006,
  -0.15121756669381203,  0.8728182536916264,  -1.53138542229656,  -3.3849589463362206,
  -0.16458774541274823,  0.9004472408314221,  -1.4091149233615692,  -3.613411396539433,
  -0.17853005970250146,  0.9253285731539653,  -1.2823613485338112,  -3.7677449288484848,
  -0.1930233856552166,  0.9473233443596456,  -1.152787735775737,  -3.8550731494912736,
  -0.2080445338422312,  0.9663251440669282,  -1.0221538280336067,  -3.876905141560477,
  -0.22356851032785197,  0.9822575621067492,  -0.8919756453390023,  -3.841196302150984,
  -0.23956877633717655,  0.9950744339966598,  -0.7637328940363731,  -3.7529273616532586,
  -0.25601750314867533,  1.004757271441477,  -0.6386597075263281,  -3.6216396854891446,
  -0.27288581932374956,  1.0113149126727092,  -0.5179751718169118,  -3.4515146572971545,
  -0.290144047881119,  1.0147799486884705,  -0.40261181076920194,  -3.2517868703465225,
  -0.3077619315444198,  1.0152074667929123,  -0.29336941610519457,  -3.030274459030009,
  -0.3257088446531329,  1.0126728162119434,  -0.1909247210281448,  -2.7929602304328944,
  -0.3439539906817817,  1.007269092079632,  -0.09580396316160587,  -2.545261937821667,
  -0.3624665847421984,  0.9991044028985486,  -0.00832380906003052,  -2.2946566462528817,
  -0.3812160207074986,  0.9883001826270612,  0.07127989502777134,  -2.045559558840712,
  -0.40017202287646336,  0.9749888581080804,  0.142876827228744,  -1.8012191796151173,
  -0.41930478233652435,  0.9593109648665744,  0.2065334827787164,  -1.5674757056852222,
  -0.43858507834823324,  0.9414148800520581,  0.2622210245501287,  -1.3419893245869465,
  -0.457984385279147,  0.9214519008647142,  0.3102840532998863,  -1.1329097280379683,
  -0.4774749656150794,  0.8995776700896025,  0.350924100520111,  -0.9395954544872396,
  -0.4970299498418205,  0.8759489986306164,  0.3844262962041706,  -0.7616394684967835,
  -0.5166234038755029,  0.8507219404641266,  0.41124723958766257,  -0.6032588735949616,
  -0.5362303849048673,  0.8240528785669349,  0.4315893465577775,  -0.4566937624927689,
  -0.5558269864455703,  0.796092385995486,  0.44621693603268325,  -0.334527600644811,
  -0.5753903734545689,  0.7669915615718423,  0.4552735940124304,  -0.22477732552396454,
  -0.594898808358532,  0.7368949154068004,  0.4592653962813549,  -0.12865546625083857,
  -0.6143316688163631,  0.7059415661087483,  0.4587541350845771,  -0.04930713070203554,
  -0.6336694580093012,  0.6742655059089414,  0.45421630136436464,  0.013351010739336425,
  -0.6528938082654727,  0.6419952540847547,  0.44601041775832667,  0.0638415188291839,
  -0.671987478773515,  0.609253193848314,  0.43444486124533294,  0.10763475557811222,
  -0.6909343480693215,  0.5761532008506691,  0.42007746549350233,  0.14051880613148526,
  -0.7097194020409532,  0.5428030801102962,  0.4033403234352994,  0.16161950988854637,
  -0.7283287180532814,  0.5093049321146808,  0.38441281350991785,  0.17945856130967625,
  -0.7467494458428221,  0.4757515272872723,  0.36382986450506455,  0.18796213055648905,
  -0.7649697857795068,  0.44223110539038724,  0.34172196901005686,  0.19507811894905616,
  -0.7829789650347543,  0.4088208614967782,  0.31885110259738225,  0.18393079031285164,
  -0.8007672122209817,  0.3755978677698798,  0.29490620241225024,  0.17747797390466974,
  -0.8183257310318415,  0.34262754617021673,  0.27035152812733393,  0.16809691827740403,
  -0.8356466733784167,  0.30997047667346334,  0.2454080646723364,  0.15613246370080863,
  -0.8527231125751265,  0.2776821034363124,  0.22007539994045897,  0.1503229051891503,
  -0.8695490170763458,  0.2458078889412257,  0.19501523005805657,  0.13197914126794016,
  -0.8861192253496697,  0.2143927998793557,  0.1700596292860306,  0.11336558889818169,
  -0.9024294224876923,  0.1834747248396055,  0.14522031860272314,  0.1008822422577469,
  -0.9184761192443098,  0.15308461722633734,  0.12088152358387703,  0.08303734747457973,
  -0.934256634298803,  0.12325141959936302,  0.09689566655697647,  0.06976962770561149,
  -0.9497690807073822,  0.09399729803877224,  0.07340833162152047,  0.059837703971412304,
  -0.9650123577317733,  0.06533743539522163,  0.05106147102606125,  0.030546304362283175,
  -0.9799861495589,  0.03729282283224224,  0.028845838288358933,  0.022613654960989917
};

}}  // namespace n2k::global_sk
